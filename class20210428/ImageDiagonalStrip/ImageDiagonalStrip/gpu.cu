#include "hip/hip_runtime.h"
#include "option.h"
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""

#ifdef PARALLEL
#define HEIGHT 1024
#define WIDTH 1024
#define CHANNELS_N 3
#define BLOCKS_N 16
#define BLOCK_THREADS_N 64
#define STRIP_WIDTH 30
#define STRIP_R 255
#define STRIP_G 0
#define STRIP_B 0

__global__ void GetDiagonallyStrippedFrameKernel(uchar* const d_pOriginalFrame, uchar* const d_pStrippedFrame)
{
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= HEIGHT) // overflow
    {
        return;
    }
    for (int j = 0; j < WIDTH; ++j)
    {
        if (i % STRIP_WIDTH == j % STRIP_WIDTH)
        {
            d_pStrippedFrame[(i * WIDTH + j) * CHANNELS_N + 0] = STRIP_B;
            d_pStrippedFrame[(i * WIDTH + j) * CHANNELS_N + 1] = STRIP_G;
            d_pStrippedFrame[(i * WIDTH + j) * CHANNELS_N + 2] = STRIP_R;
        }
        else
        {
            for (int k = 0; k < CHANNELS_N; ++k)
            {
                d_pStrippedFrame[(i * WIDTH + j) * CHANNELS_N + k] = d_pOriginalFrame[(i * WIDTH + j) * CHANNELS_N + k];
            }
        }
    }
}

int main()
{
    // read image
    cv::Mat kCar = cv::imread("car.jpg");

    // start time recorder
    hipEvent_t kStart;
    hipEvent_t kStop;
    hipEventCreate(&kStart);
    hipEventCreate(&kStop);
    hipEventRecord(kStart, 0);

    // get original frame
    const unsigned int FRAME_SIZE = kCar.rows * kCar.step;
    uchar* const pFrame = (uchar*)malloc(FRAME_SIZE * sizeof(uchar));
    for (int i = 0; i < HEIGHT; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            for (int k = 0; k < CHANNELS_N; ++k)
            {
                pFrame[(i * WIDTH + j) * CHANNELS_N + k] = kCar.at<cv::Vec3b>(i, j)[k];
            }
        }
    }

    // CUDA prepare [d]evice frames
    uchar* d_pOriginalFrame;
    uchar* d_pStrippedFrame;
    hipMalloc((void**)&d_pOriginalFrame, FRAME_SIZE * sizeof(uchar));
    hipMalloc((void**)&d_pStrippedFrame, FRAME_SIZE * sizeof(uchar));
    hipMemcpy(d_pOriginalFrame, pFrame, FRAME_SIZE * sizeof(uchar), hipMemcpyHostToDevice);

    // CUDA get [d]evice stripped frame
    GetDiagonallyStrippedFrameKernel<<<BLOCKS_N, BLOCK_THREADS_N>>>(d_pOriginalFrame, d_pStrippedFrame);

    // CUDA write result onto frame
    hipMemcpy(pFrame, d_pStrippedFrame, FRAME_SIZE * sizeof(uchar), hipMemcpyDeviceToHost);

    // CUDA free [d]evice frames
    hipFree(d_pOriginalFrame);
    hipFree(d_pStrippedFrame);

    // load frame to image
    for (int i = 0; i < HEIGHT; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            for (int k = 0; k < CHANNELS_N; ++k)
            {
                kCar.at<cv::Vec3b>(i, j)[k] = pFrame[(i * WIDTH + j) * CHANNELS_N + k];
            }
        }
    }

    // free frame
    free(pFrame);

    // stop time recorder
    hipEventRecord(kStop, 0);
    hipEventSynchronize(kStop);
    float fTimeMs = 0.f;
    hipEventElapsedTime(&fTimeMs, kStart, kStop);
    hipEventDestroy(kStart);
    hipEventDestroy(kStop);
    printf("Process data took me %f milliseconds.\n", fTimeMs);

    // show image
    cv::imshow("Image Horizontal Flip", kCar);
    cv::waitKey(0);

    return 0;
}
#endif