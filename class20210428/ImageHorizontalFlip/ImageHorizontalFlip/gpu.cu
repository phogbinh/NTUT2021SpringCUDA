#include "hip/hip_runtime.h"
#include "option.h"
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""

#ifdef PARALLEL
#define HEIGHT 1024
#define WIDTH 1024
#define CHANNELS_N 3
#define BLOCKS_N 16
#define BLOCK_THREADS_N 64

__global__ void GetHorizontallyFlippedFrameKernel(uchar* const d_pOriginalFrame, uchar* const d_pFlippedFrame)
{
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= HEIGHT) // overflow
    {
        return;
    }
    for (int j = 0; j < WIDTH; ++j)
    {
        for (int k = 0; k < CHANNELS_N; ++k)
        {
            d_pFlippedFrame[(i * WIDTH + j) * CHANNELS_N + k] = d_pOriginalFrame[(i * WIDTH + (WIDTH - 1 - j)) * CHANNELS_N + k];
        }
    }
}


int main()
{
    // read image
    cv::Mat kCar = cv::imread("car.jpg");

    // start time recorder
    hipEvent_t kStart;
    hipEvent_t kStop;
    hipEventCreate(&kStart);
    hipEventCreate(&kStop);
    hipEventRecord(kStart, 0);

    // get original frame
    const unsigned int FRAME_SIZE = kCar.rows * kCar.step;
    uchar* const pFrame = (uchar*)malloc(FRAME_SIZE * sizeof(uchar));
    for (int i = 0; i < HEIGHT; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            for (int k = 0; k < CHANNELS_N; ++k)
            {
                pFrame[(i * WIDTH + j) * CHANNELS_N + k] = kCar.at<cv::Vec3b>(i, j)[k];
            }
        }
    }

    // CUDA prepare [d]evice frames
    uchar* d_pOriginalFrame;
    uchar* d_pFlippedFrame;

    hipMalloc((void**)&d_pOriginalFrame, FRAME_SIZE * sizeof(uchar));
    hipMalloc((void**)&d_pFlippedFrame,  FRAME_SIZE * sizeof(uchar));

    hipMemcpy(d_pOriginalFrame, pFrame, FRAME_SIZE * sizeof(uchar), hipMemcpyHostToDevice);

    // CUDA get [d]evice flipped frame
    GetHorizontallyFlippedFrameKernel<<<BLOCKS_N, BLOCK_THREADS_N>>>(d_pOriginalFrame, d_pFlippedFrame);

    // CUDA write result onto frame
    hipMemcpy(pFrame, d_pFlippedFrame, FRAME_SIZE * sizeof(uchar), hipMemcpyDeviceToHost);

    // CUDA free [d]evice frames
    hipFree(d_pOriginalFrame);
    hipFree(d_pFlippedFrame);

    // load frame to image
    for (int i = 0; i < HEIGHT; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            for (int k = 0; k < CHANNELS_N; ++k)
            {
                kCar.at<cv::Vec3b>(i, j)[k] = pFrame[(i * WIDTH + j) * CHANNELS_N + k];
            }
        }
    }

    // free frame
    free(pFrame);

    // stop time recorder
    hipEventRecord(kStop, 0);
    hipEventSynchronize(kStop);
    float fTimeMs = 0.f;
    hipEventElapsedTime(&fTimeMs, kStart, kStop);
    hipEventDestroy(kStart);
    hipEventDestroy(kStop);
    printf("Process data took me %f milliseconds.\n", fTimeMs);

    // show image
    cv::imshow("Image Horizontal Flip", kCar);
    cv::waitKey(0);

    return 0;
}
#endif