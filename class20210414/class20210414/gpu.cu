#include "hip/hip_runtime.h"
#include "option.h"
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""

#define BLOCKS_N 32
#define BLOCK_THREADS_N 64

#ifdef PARALLEL
__global__ void ProcessDataKernel(const uchar* const d_pFlowerData,
                                  const uchar* const d_pCarData,
                                  const int HEIGHT,
                                  const int WIDTH,
                                  const int CHANNELS_N,
                                  uchar* const d_pData)
{
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= HEIGHT) // overflow
    {
        return;
    }
    for (int j = 0; j < WIDTH; ++j)
    {
        for (int k = 0; k < CHANNELS_N; ++k)
        {
            const int nIndex = (i * WIDTH + j) * CHANNELS_N + k;
            d_pData[nIndex] = (j < WIDTH / 2) ? d_pFlowerData[nIndex] : d_pCarData[nIndex];
        }
    }
}

int main()
{
    // read image
    cv::Mat kFlower = cv::imread("flower.jpg");
    cv::Mat kCar = cv::imread("car.jpg");

    // get image data
    const int HEIGHT = kFlower.rows;
    const int WIDTH = kFlower.cols;
    const int CHANNELS_N = kFlower.channels();
    const int DATA_SIZE = HEIGHT * WIDTH * kFlower.elemSize();
    uchar* const pFlowerData = (uchar*)kFlower.data;
    const uchar* const pCarData = (uchar*)kCar.data;

    // start time recorder
    hipEvent_t kStart;
    hipEvent_t kStop;
    hipEventCreate(&kStart);
    hipEventCreate(&kStop);
    hipEventRecord(kStart, 0);

    // CUDA prepare [d]evice data
    uchar* d_pFlowerData;
    uchar* d_pCarData;
    uchar* d_pData;

    hipMalloc((void**)&d_pFlowerData, DATA_SIZE * sizeof(uchar));
    hipMalloc((void**)&d_pCarData,    DATA_SIZE * sizeof(uchar));
    hipMalloc((void**)&d_pData,       DATA_SIZE * sizeof(uchar));

    hipMemcpy(d_pFlowerData, pFlowerData, DATA_SIZE * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(d_pCarData,    pCarData,    DATA_SIZE * sizeof(uchar), hipMemcpyHostToDevice);

    // CUDA process data
    ProcessDataKernel<<<BLOCKS_N, BLOCK_THREADS_N>>>(d_pFlowerData, d_pCarData, HEIGHT, WIDTH, CHANNELS_N, d_pData);

    // CUDA write result onto flower image
    hipMemcpy(pFlowerData, d_pData, DATA_SIZE * sizeof(uchar), hipMemcpyDeviceToHost);

    // CUDA free [d]evice data
    hipFree(d_pFlowerData);
    hipFree(d_pCarData);
    hipFree(d_pData);

    // stop time recorder
    hipEventRecord(kStop, 0);
    hipEventSynchronize(kStop);
    float fTimeMs = 0.f;
    hipEventElapsedTime(&fTimeMs, kStart, kStop);
    hipEventDestroy(kStart);
    hipEventDestroy(kStop);
    printf("Process data took me %f milliseconds.\n", fTimeMs);

    // show image
    cv::imshow("Class 20210414", kFlower);
    cv::waitKey(0);

    return 0;
}
#endif
